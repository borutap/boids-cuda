#include "hip/hip_runtime.h"
#include "boids_common.h"
#include "boids_gpu.h"
#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"

__global__ void kernel_test(Boid *boids, glm::mat4 *trans, int n,
                            float centering_factor, float visual_range,
                            float margin, float turn_factor,
                            float speed_limit, float min_distance,
                            float avoid_factor, float matching_factor,
                            float mouseX, float mouseY)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n)
    {
        return;
    }
        
    fly_towards_center(boids, index, n, centering_factor, visual_range);
    avoid_others(boids, index, n, min_distance, avoid_factor);    
    // zeby zmiana predkosci (dx, dy) w avoid_others 
    // nie zaburzyla sredniej liczonej
    // w innym watku w match_velocity
    __syncthreads();
    match_velocity(boids, index, n, matching_factor, visual_range);
    if (mouseX != -2)
        avoid_mouse(boids, index, mouseX, mouseY);
    limit_speed(boids, index, speed_limit);
    keep_within_bounds(boids, index, margin, turn_factor);
    
    Boid &boid = boids[index];
    boid.x += boid.dx;
    boid.y += boid.dy;
    float angle = glm::atan(boid.dy / boid.dx);
    float pi = glm::pi<float>();
    if (boid.dx <= 0)
    {
        angle += pi / 2;
    }        
    else
    {
        angle -= pi / 2;
    }
    auto transformation = glm::translate(glm::mat4(1.0f), glm::vec3(boid.x, boid.y, 0.0f));
    transformation = glm::rotate(transformation, angle, glm::vec3(0.0f, 0.0f, 1.0f));
    trans[index] = transformation;
}

__device__ void fly_towards_center(Boid *boids, int index, int n,
                                   float centering_factor,
                                   float visual_range)
{    
    Boid &boid = boids[index];
    // centering_factor = adjust velocity by this %

    float centerX = 0.0f;
    float centerY = 0.0f;
    int num_neighbors = 0;

    for (unsigned int i = 0; i < n; i++)
    {
        Boid &other = boids[i];
        if (distance(boid, other) < visual_range)
        {
            centerX += other.x;
            centerY += other.y;
            num_neighbors += 1;
        }
    }

    if (num_neighbors)
    {
        centerX = centerX / num_neighbors;
        centerY = centerY / num_neighbors;

        boid.dx += (centerX - boid.x) * centering_factor;
        boid.dy += (centerY - boid.y) * centering_factor;
    }
}

__device__ float distance(Boid &boid1, Boid &boid2)
{
    return glm::sqrt(
        (boid1.x - boid2.x) * (boid1.x - boid2.x) +
            (boid1.y - boid2.y) * (boid1.y - boid2.y)
    );
}

__device__ void keep_within_bounds(Boid *boids, int index,
                                   float margin, float turn_factor)
{
    Boid &boid = boids[index];
    // const float margin = 0.1f;
    // const float turn_factor = 1.0f / 2000;

    if (boid.x < -1.0f + margin)
        boid.dx += turn_factor;    

    if (boid.x > 1.0f - margin)
        boid.dx -= turn_factor;

    if (boid.y < -1.0f + margin)
        boid.dy += turn_factor;    

    if (boid.y > 1.0f - margin)
        boid.dy -= turn_factor;
}

__device__ void limit_speed(Boid *boids, int index, float speed_limit)
{
    Boid &boid = boids[index];

    float speed = glm::sqrt(boid.dx * boid.dx + boid.dy * boid.dy);
    if (speed <= speed_limit)
        return;
    boid.dx = (boid.dx / speed) * speed_limit;
    boid.dy = (boid.dy / speed) * speed_limit;
}

__device__ void avoid_others(Boid *boids, int index, int n,
                             float min_distance, float avoid_factor)
{
    Boid &boid = boids[index];
    // min_distance = The distance to stay away from other boids
    // avoid_factor = Adjust velocity by this %
    float moveX = 0;
    float moveY = 0;
    for (unsigned int i = 0; i < n; i++)
    {
        if (i == index)
            continue;
        Boid &other = boids[i];        
        if (distance(boid, other) < min_distance)
        {
            moveX += boid.x - other.x;
            moveY += boid.y - other.y;
        }
    } 
    
    boid.dx += moveX * avoid_factor;
    boid.dy += moveY * avoid_factor;
}

__device__ void avoid_mouse(Boid *boids, int index, float mouseX, float mouseY)
{
    Boid &boid = boids[index];
    // min_distance = The distance to stay away from other boids
    // avoid_factor = Adjust velocity by this %
    // na razie na stale
    float min_distance = 0.07;
    float avoid_factor = 0.5;    

    if (glm::sqrt(
            (boid.x - mouseX) * (boid.x - mouseX) +
            (boid.y - mouseY) * (boid.y - mouseY)) < min_distance)
    {
        boid.dx += boid.x - mouseX * avoid_factor;
        boid.dy += boid.y - mouseY * avoid_factor;        
    }        
}

__device__ void match_velocity(Boid *boids, int index, int n,
                               float matching_factor,
                               float visual_range)
{
    Boid &boid = boids[index];

    float avgDX = boid.dx;
    float avgDY = boid.dy;
    int num_neighbors = 0;

    for (unsigned int i = 0; i < n; i++)
    {
        if (i == index)
            continue;
        Boid &other = boids[i];
        if (distance(boid, other) < visual_range)
        {
            avgDX += other.dx;
            avgDY += other.dy;
            num_neighbors += 1;
        }
    }
    // zeby zmiana dx, dy nizej nie zaburzyla
    // sredniej liczonej w innym watku wyzej
    __syncthreads();    
    if (num_neighbors)
    {
        avgDX = avgDX / num_neighbors;
        avgDY = avgDY / num_neighbors;

        boid.dx += (avgDX - boid.dx) * matching_factor;
        boid.dy += (avgDY - boid.dy) * matching_factor;
    }
}

void copy_boid_structure_to_device(Boid **boids, Boid **d_pointer, int n)
{
    size_t size = sizeof(Boid);
    hipMalloc(d_pointer, n * size);
    hipMemcpy(*d_pointer, *boids, n * size, hipMemcpyHostToDevice);
}

void copy_trans_matrix_to_device(glm::mat4 **mat, glm::mat4 **d_mat, int n)
{
    size_t size = sizeof(glm::mat4);
    hipMalloc(d_mat, n * size);
    hipMemcpy(*d_mat, *mat, n * size, hipMemcpyHostToDevice);
}

void copy_trans_matrix_to_host(glm::mat4 **mat, glm::mat4 **d_mat, int n)
{
    hipMemcpy(*mat, *d_mat, n *  sizeof(glm::mat4), hipMemcpyDeviceToHost);
}

void copy_boid_structure_to_host(Boid **boids, Boid **d_pointer, int n)
{   
    hipMemcpy(*boids, *d_pointer, n *  sizeof(Boid), hipMemcpyDeviceToHost);
}