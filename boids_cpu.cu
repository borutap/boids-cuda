#include "hip/hip_runtime.h"
#include "boids_common.h"
#include "boids_cpu.h"
#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"

void cpu::cpu_test(Boid *boids, glm::mat4 *trans, int n,
                   float centering_factor, float visual_range,
                   float margin, float turn_factor,
                   float speed_limit, float min_distance,
                   float avoid_factor, float matching_factor)
{
    for (int index = 0; index < n; index++)
    {
        fly_towards_center(boids, index, n,
                           centering_factor, visual_range);
        avoid_others(boids, index, n, min_distance, avoid_factor);
                
        match_velocity(boids, index, n, matching_factor, visual_range);
        limit_speed(boids, index, speed_limit);
        keep_within_bounds(boids, index, margin, turn_factor);    
        Boid &boid = boids[index];
        boid.x += boid.dx;
        boid.y += boid.dy;
        float angle = glm::atan(boid.dy / boid.dx);
        float pi = glm::pi<float>();
        if (boid.dx <= 0)
        {
            angle += pi / 2;
        }        
        else
        {
            angle -= pi / 2;
        }
        auto transformation = glm::translate(glm::mat4(1.0f), glm::vec3(boid.x, boid.y, 0.0f));
        transformation = glm::rotate(transformation, angle, glm::vec3(0.0f, 0.0f, 1.0f));
        trans[index] = transformation;
    }        
}

void cpu::fly_towards_center(Boid *boids, int index, int n,
                            float centering_factor,
                            float visual_range)
{    
    Boid &boid = boids[index];

    float centerX = 0.0f;
    float centerY = 0.0f;
    int num_neighbors = 0;

    for (int i = 0; i < n; i++)
    {
        Boid &other = boids[i];
        if (distance(boid, other) < visual_range)
        {
            centerX += other.x;
            centerY += other.y;
            num_neighbors += 1;
        }
    }

    if (num_neighbors)
    {
        centerX = centerX / num_neighbors;
        centerY = centerY / num_neighbors;

        boid.dx += (centerX - boid.x) * centering_factor;
        boid.dy += (centerY - boid.y) * centering_factor;
    }
}

float cpu::distance(Boid &boid1, Boid &boid2)
{
    return glm::sqrt(
        (boid1.x - boid2.x) * (boid1.x - boid2.x) +
            (boid1.y - boid2.y) * (boid1.y - boid2.y)
    );
}

void cpu::keep_within_bounds(Boid *boids, int index,
                            float margin, float turn_factor)
{
    Boid &boid = boids[index];

    if (boid.x < -1.0f + margin)
        boid.dx += turn_factor;    

    if (boid.x > 1.0f - margin)
        boid.dx -= turn_factor;

    if (boid.y < -1.0f + margin)
        boid.dy += turn_factor;    

    if (boid.y > 1.0f - margin)
        boid.dy -= turn_factor;
}

void cpu::limit_speed(Boid *boids, int index, float speed_limit)
{
    Boid &boid = boids[index];

    float speed = glm::sqrt(boid.dx * boid.dx + boid.dy * boid.dy);
    if (speed <= speed_limit)
        return;
    boid.dx = (boid.dx / speed) * speed_limit;
    boid.dy = (boid.dy / speed) * speed_limit;
}

void cpu::avoid_others(Boid *boids, int index, int n,
                      float min_distance, float avoid_factor)
{
    Boid &boid = boids[index];
    
    float moveX = 0;
    float moveY = 0;
    for (unsigned int i = 0; i < n; i++)
    {
        if (i == index)
            continue;
        Boid &other = boids[i];        
        if (distance(boid, other) < min_distance)
        {
            moveX += boid.x - other.x;
            moveY += boid.y - other.y;
        }
    } 
    
    boid.dx += moveX * avoid_factor;
    boid.dy += moveY * avoid_factor;
}

void cpu::match_velocity(Boid *boids, int index, int n,
                        float matching_factor,
                        float visual_range)
{
    Boid &boid = boids[index];

    float avgDX = boid.dx;
    float avgDY = boid.dy;
    int num_neighbors = 0;

    for (int i = 0; i < n; i++)
    {
        if (i == index)
            continue;
        Boid &other = boids[i];
        if (distance(boid, other) < visual_range)
        {
            avgDX += other.dx;
            avgDY += other.dy;
            num_neighbors += 1;
        }
    }
           
    if (num_neighbors)
    {
        avgDX = avgDX / num_neighbors;
        avgDY = avgDY / num_neighbors;

        boid.dx += (avgDX - boid.dx) * matching_factor;
        boid.dy += (avgDY - boid.dy) * matching_factor;
    }
}